#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void selectionSort(int *arr, int n) {
    int i, j, minIdx;
    for (i = 0; i < n - 1; i++) {
        minIdx = i;
        for (j = i + 1; j < n; j++) {
            if (arr[j] < arr[minIdx]) {
                minIdx = j;
            }
        }
        int temp = arr[i];
        arr[i] = arr[minIdx];
        arr[minIdx] = temp;
    }
}
int main() {
    int n;
    printf("Enter number of elements: ");
    scanf("%d", &n);
    int h_arr[n];
    printf("Enter elements of array: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &h_arr[i]);
    }
    int *d_arr;
    hipMalloc((void**)&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, h_arr, n * sizeof(int), hipMemcpyHostToDevice);
    selectionSort<<<1, 1>>>(d_arr, n);
    hipMemcpy(h_arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Sorted Array: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");
    return 0;
}