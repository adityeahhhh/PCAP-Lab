#include <stdio.h>
#include <hip/hip_runtime.h>

#define MAX_ITEMS 10
#define MAX_FRIENDS 100

typedef struct {
    char name[50];
    float price;
} Item;

__global__ void calculateTotal(float *prices, int *quantities, float *totals, int numItems, int numFriends) {
    int friendIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (friendIndex < numFriends) {
        float total = 0.0f;
        
        for (int i = 0; i < numItems; i++) {
            int qty = quantities[friendIndex * numItems + i];
            if (qty < 0) {
                printf("Error: Negative quantity detected for Friend %d, Item %d\n", friendIndex, i);
            }
            total += prices[i] * qty;
        }
        
        totals[friendIndex] = total;
    }
}

int main() {
    Item items[MAX_ITEMS] = {
        {"Shirt", 20.0},
        {"Jeans", 40.0},
        {"Shoes", 50.0},
        {"Hat", 15.0},
        {"Sunglasses", 30.0},
        {"Bag", 60.0},
        {"Watch", 100.0},
        {"Jacket", 80.0},
        {"Scarf", 25.0},
        {"Wallet", 40.0}
    };

    int numItems = MAX_ITEMS;
    int numFriends;

    printf("Enter number of friends (N): ");
    scanf("%d", &numFriends);

    int quantities[MAX_FRIENDS * MAX_ITEMS];
    float totals[MAX_FRIENDS];

    for (int i = 0; i < numFriends; i++) {
        printf("\nFriend %d, please enter the quantities of the following items:\n", i + 1);
        for (int j = 0; j < numItems; j++) {
            printf("%s (Price: %.2f): ", items[j].name, items[j].price);
            scanf("%d", &quantities[i * numItems + j]);
        }
    }

    float prices[MAX_ITEMS];
    for (int i = 0; i < numItems; i++) {
        prices[i] = items[i].price;
    }

    float *d_prices, *d_totals;
    int *d_quantities;

    hipMalloc((void**)&d_prices, numItems * sizeof(float));
    hipMalloc((void**)&d_quantities, numFriends * numItems * sizeof(int));
    hipMalloc((void**)&d_totals, numFriends * sizeof(float));

    hipMemcpy(d_prices, prices, numItems * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_quantities, quantities, numFriends * numItems * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 32;
    int gridSize = (numFriends + blockSize - 1) / blockSize;
    calculateTotal<<<gridSize, blockSize>>>(d_prices, d_quantities, d_totals, numItems, numFriends);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(error));
        return -1;
    }

    hipMemcpy(totals, d_totals, numFriends * sizeof(float), hipMemcpyDeviceToHost);

    float grandTotal = 0.0f;
    for (int i = 0; i < numFriends; i++) {
        printf("\nFriend %d's total purchase: $%.2f\n", i + 1, totals[i]);
        grandTotal += totals[i];
    }

    printf("\nGrand Total for all friends: $%.2f\n", grandTotal);

    hipFree(d_prices);
    hipFree(d_quantities);
    hipFree(d_totals);

    return 0;
}
